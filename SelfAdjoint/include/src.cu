#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "helper.h"

#include <Eigen/Dense>
#include <Eigen/Eigenvalues>

__global__ void kernel()
{
    using MatT = Eigen::Matrix3f;

    MatT H;
    H << 1, 0, 0,  //
        0, 2, 0,   //
        0, 0, 3;

    Eigen::SelfAdjointEigenSolver<MatT> eig(H);

    printf("\n Eigenvalues: %f, %f, %f\n ",
           eig.eigenvalues()[0],
           eig.eigenvalues()[1],
           eig.eigenvalues()[2]);

    MatT D = eig.eigenvalues().asDiagonal();

    printf("\n Eigenvalues as diag: %f, %f, %f\n ", D(0, 0), D(1, 1), D(2, 2));
}


int main(int argc, char** argv)
{
    kernel<<<1, 1>>>();

    CUDA_ERROR(hipDeviceSynchronize());

    return 0;
}
